
#include <hip/hip_runtime.h>
#define RADIUS 25

__device__ unsigned int GetPixelWithPadding(unsigned int x, unsigned int y, unsigned int width, unsigned int padding)
{
    return (x + padding + (y + padding) * (width + 2 * padding)) * gridDim.z + blockIdx.z;
}

__device__ unsigned int GetPixel(unsigned int x, unsigned int y, unsigned int width)
{
    return GetPixelWithPadding(x, y, width, 0);
}

// radius can't be an unsigned int because "-radius" is used in the for...loops
extern "C" __global__ void BoxBlur(int *in_array, int *out_array, unsigned int width, unsigned int height)
{
    unsigned int denominator = 0;
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int index = GetPixel(x, y, width);
    out_array[index] = 0;
#pragma unroll
    for (int dx = -RADIUS; dx <= RADIUS; ++dx)
    {
#pragma unroll
        for (int dy = -RADIUS; dy <= RADIUS; ++dy)
        {
            const int neighborX = x + dx;
            const int neighborY = y + dy;
            const int neighborIndex = GetPixelWithPadding(neighborX, neighborY, width, RADIUS);
            out_array[index] += in_array[neighborIndex];
            ++denominator;
        }
    }
    out_array[index] /= denominator;
}